
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
using namespace std;


__global__ void MatMul(double * A, double * B ,double * C,int m,int n,int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    if (i < m && j < k){
        for(int x = 0;x < n;x++){
            sum += A[i * n + x] * B[x * k + j];
        }
        C[i * k + j] = sum;
    }
}

void CPU_MatMul(double * A, double * B ,double * C,int m,int n,int k){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < k;j++){
            for(int x = 0;x < n;x++){
                C[i * k + j] += A[i * n + x] * B[x * k + j];
            }
        }
    }
}
int main()
{
    timeval t1, t2;
    int x,y;
    cout << "Input threadsPerBlock.x:";
    cin >> x;
    cout << "Input threadsPerBlock.y:";
    cin >> y;
    dim3 threadsPerBlock(x,y);
    int m, n, k;
    cout << "Input problem size:";
    cin >> m;
    n = m;
    k = m;

    dim3 numBlocks((m % threadsPerBlock.x) ? m / threadsPerBlock.x + 1 : m / threadsPerBlock.x ,(k % threadsPerBlock.y) ? k / threadsPerBlock.y + 1 : k / threadsPerBlock.y);
    double *A,*B,*C,*C1;
    A = (double*)malloc(sizeof(double) * m * n);
    B = (double*)malloc(sizeof(double) * k * n);
    C = (double*)malloc(sizeof(double) * m * k);
    C1 = (double*)malloc(sizeof(double) * m * k);
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            A[i * n + j] = rand() % 10;
        }
    }
    for(int i = 0;i < n;i++){
        for(int j = 0;j < k;j++){
            B[i * k + j] = rand() % 10;
        }
    }
    memset(C,0,sizeof(C));
    int omp_threads;
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    cout << "Input number of omp threads:";
    cin >> omp_threads;
    gettimeofday(&t1, NULL);
    #pragma omp parallel num_threads(omp_threads)
    {
        int id = omp_get_thread_num();
        int size = omp_get_num_threads();
        hipSetDevice(id % gpu_count);
        double * d_A,*d_B,*d_C;
        hipMalloc(&d_A, sizeof(double) * m * n / size);
        hipMalloc(&d_B,sizeof(double) * n * k);
        hipMalloc(&d_C,sizeof(double) * m * k / size);

        hipMemcpy(d_A, A + id * m * n / size, sizeof(double) * m * n / size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, sizeof(double) * n * k, hipMemcpyHostToDevice);
        MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m / size, n, k);
        hipMemcpy(C + id * m * k / size, d_C, sizeof(double) * m * k / size, hipMemcpyDeviceToHost);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    gettimeofday(&t2, NULL);
    printf("GPU time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);
    
    gettimeofday(&t1, NULL);
    CPU_MatMul(A,B,C1,m,n,k);
    gettimeofday(&t2, NULL);
    printf("CPU time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);

    int flag = 0;
    for(int i = 0;i < m * k;i++){
        if(fabs((C[i] - C1[i])) > 1e-4){
            flag = 1;
            break;
        }
    }
    if(flag){
        cout << "Wrong result." << endl;
    }
    else {
        cout << "The results are correct. " << endl;
    }
  
    free(A);
    free(B);
    free(C);
    free(C1);
}
