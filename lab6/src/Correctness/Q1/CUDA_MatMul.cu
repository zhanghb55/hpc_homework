
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
using namespace std;


__global__ void MatMul(double * A, double * B ,double * C,int m,int n,int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double sum = 0;
    if (i < m && j < k){
        for(int x = 0;x < n;x++){
            sum += A[i * n + x] * B[x * k + j];
	    }
    	C[i * k + j] = sum;
    }
}

void CPU_MatMul(double * A, double * B ,double * C,int m,int n,int k){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < k;j++){
            for(int x = 0;x < n;x++){
                C[i * k + j] += A[i * n + x] * B[x * k + j];
            }
        }
    }
}
int main()
{
    timeval t1, t2;
    int x,y;
    cout << "Input threadsPerBlock.x:";
    cin >> x;
    cout << "Input threadsPerBlock.y:";
    cin >> y;
    dim3 threadsPerBlock(x,y);
    int m, n, k;
    cout << "Input problem size:";
    cin >> m;
    n = m;
    k = m;

    dim3 numBlocks((m % threadsPerBlock.x) ? m / threadsPerBlock.x + 1 : m / threadsPerBlock.x ,(k % threadsPerBlock.y) ? k / threadsPerBlock.y + 1 : k / threadsPerBlock.y);
    double *A,*B,*C,*C1;
    A = (double*)malloc(sizeof(double) * m * n);
    B = (double*)malloc(sizeof(double) * k * n);
    C = (double*)malloc(sizeof(double) * m * k);
    C1 = (double*)malloc(sizeof(double) * m * k);
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            A[i * n + j] = rand() % 10;
        }
    }
    for(int i = 0;i < n;i++){
        for(int j = 0;j < k;j++){
            B[i * k + j] = rand() % 10;
        }
    }
    memset(C,0,sizeof(C));
    memset(C1,0,sizeof(C));
    double * d_A,*d_B,*d_C;
    gettimeofday(&t1, NULL);
    hipMalloc(&d_A, sizeof(double) * m * n);
    hipMalloc(&d_B,sizeof(double) * n * k);
    hipMalloc(&d_C,sizeof(double) * m * k);
    hipMemcpy(d_A, A, sizeof(double) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(double) * n * k, hipMemcpyHostToDevice);
    MatMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);
    hipMemcpy(C, d_C, sizeof(double) * m * k, hipMemcpyDeviceToHost);
    gettimeofday(&t2, NULL);
    printf("GPU time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);
    gettimeofday(&t1, NULL);
    CPU_MatMul(A,B,C1,m,n,k);
    gettimeofday(&t2, NULL);
    printf("CPU time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);

    int flag = 0;
    for(int i = 0;i < m * k;i++){
        if(fabs((C[i] - C1[i])) > 1e-4){
            flag = 1;
            break;
        }
    }
    if(flag){
        cout << "Wrong result." << endl;
    }
    else {
        cout << "The results are correct. " << endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    free(C1);
}