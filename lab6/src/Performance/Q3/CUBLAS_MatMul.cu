#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
using namespace std;

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void CPU_MatMul(double * A, double * B ,double * C,int m,int n,int k){
    for(int i = 0;i < m;i++){
        for(int j = 0;j < k;j++){
            for(int x = 0;x < n;x++){
                C[IDX2C(i,j,k)] += A[i * n + x] * B[x * k + j];
            }
        }
    }
}
int main()
{
    int m,n,k;
    timeval t1, t2;
    cout << "Input problem size:";
    cin >> m;
    n = m;
    k = m;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double *A,*B,*C;
    A = (double*)malloc(sizeof(double) * m * n);
    B = (double*)malloc(sizeof(double) * k * n);
    C = (double*)malloc(sizeof(double) * m * k);
    
    for(int i = 0;i < m;i++){
        for(int j = 0;j < n;j++){
            A[i * n + j] = rand() % 10;
        }
    }
    for(int i = 0;i < n;i++){
        for(int j = 0;j < k;j++){
            B[i * k + j] = rand() % 10;
        }
    }
    
    memset(C,0,sizeof(C));

    double * d_A,*d_B,*d_C;
    gettimeofday(&t1, NULL);
    hipMalloc(&d_A, sizeof(double) * m * n);
    hipMalloc(&d_B,sizeof(double) * n * k);
    hipMalloc(&d_C,sizeof(double) * m * k);
    hipMemcpy(d_A, A, sizeof(double) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(double) * n * k, hipMemcpyHostToDevice);
    gettimeofday(&t1, NULL);
    double a = 1,b = 0;
    hipblasDgemm( 
            handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T,   
            m, n, k,
            &a, //alpha
            d_A, n, 
            d_B, k,         
            &b, //beta        
            d_C, m
    );

    hipMemcpy(C, d_C, sizeof(double) * m * k, hipMemcpyDeviceToHost);
    gettimeofday(&t2, NULL);
    printf("GPU time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);




    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    hipblasDestroy(handle);
}

