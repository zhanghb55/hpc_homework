
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
using namespace std;
#define IDX2C(i,j,ld) (((i)*(ld))+(j))

__global__ void convolution(float * mat,float * filter,float * res,
                            int height_stride,int width_stride,
                            int mat_height,int mat_width,
                            int filter_height,int filter_width,
                            int res_height,int res_width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;
    if(i < res_height && j < res_width){
        for(int x = 0;x < filter_height;x++){
            for(int y = 0;y < filter_width;y++){
                sum += mat[IDX2C(i * height_stride + x,j * width_stride + y,mat_width)] * filter[IDX2C(x,y,filter_width)];
            }
        }
        *(res + IDX2C(i,j,res_width)) += sum; 
    }
}


int main()
{
    timeval t1, t2;
    int x,y;    
    cout << "Input threadsPerBlock.x:";
    cin >> x;
    cout << "Input threadsPerBlock.y:";
    cin >> y;
    dim3 threadsPerBlock(x,y);
    int height, width;
    cout << "Input problem size:";
    cin >> height;
    width = height;
    int filter_height = 3;
    int filter_width = 3;
    
    int channel = 3;
    int stride;
    cout << "Input stride:";
    cin >> stride;
    int padding = ((((height - filter_height) / stride + 1) * stride - (height - filter_height)) % stride ) / 2;
    float* Mat[channel];
    float* filter[channel];

    int Mat_size = sizeof(float) * (height + 2 * padding) * (width + 2 * padding);
    int res_size = sizeof(float) * ((height - filter_height + 2 * padding) / stride + 1) * ((width - filter_width + 2 * padding) / stride + 1);
    int filter_size = sizeof(float) * filter_height * filter_width;

    for(int k = 0;k < channel;k++){
        Mat[k] = (float*) malloc(Mat_size);
        memset(Mat[k],0,sizeof(Mat[k]));
        for(int i = padding;i < height + padding;i++){
            for(int j = padding;j < width + padding;j++){
                Mat[k][IDX2C(i,j,width + 2 * padding)] = IDX2C(i,j,width + 2 * padding);
            }
        }
    }
    for(int i = 0;i < channel;i++){
        filter[i] = (float*)malloc(filter_size);
        for(int j = 0;j < filter_height * filter_width;j++){
            filter[i][j] = j + 1;
        }
    }
    float * res;
    res = (float*)malloc(res_size);

    gettimeofday(&t1, NULL);
    float * d_Mat[channel], *d_filter[channel];
    float * final_res;
    hipMalloc(&final_res,res_size);
    hipMemset(final_res,0,res_size);

    for(int i = 0;i < channel;i++){
        hipMalloc(&d_Mat[i], Mat_size);
        hipMemcpy(d_Mat[i],Mat[i],Mat_size,hipMemcpyHostToDevice);
    }

    for(int i = 0;i < channel;i++){
        hipMalloc(&d_filter[i],filter_size);
        hipMemcpy(d_filter[i],filter[i],filter_size,hipMemcpyHostToDevice);
    }

    int res_height = (height - filter_height + 2 * padding) / stride + 1;
    int res_width = (width - filter_width + 2 * padding) / stride + 1;
    dim3 numBlocks((res_height % threadsPerBlock.x) ? res_height / threadsPerBlock.x + 1 : res_height / threadsPerBlock.x ,(res_width % threadsPerBlock.y) ? res_width / threadsPerBlock.y + 1 : res_width / threadsPerBlock.y);

    for(int i = 0;i < channel;i++){
        convolution<<<numBlocks, threadsPerBlock>>>(d_Mat[i],d_filter[i],final_res,stride,stride,height + 2 * padding,width + 2 * padding,filter_height,filter_width,res_height,res_width);
    }
    gettimeofday(&t2, NULL);
    printf("convolution time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);

    hipMemcpy(res,final_res,res_size,hipMemcpyDeviceToHost);

    /*
    for(int x = 0;x < channel;x++){
        cout << "Input channel " << x << " after padding:" << endl;
        for(int i = 0;i < height + 2 * padding;i++){
            for(int j = 0;j < width + 2 * padding;j++){
                cout << Mat[x][i * (width + 2 * padding) + j] << " ";
            }
            cout << endl;
        }
    }

    cout << endl;

    for(int x = 0;x < channel;x++){
        cout << "Filter channel " << x << ":" << endl;
        for(int i = 0;i < filter_height;i++){
            for(int j = 0;j < filter_width;j++){
                cout << filter[x][i * filter_width + j] << ' ';
            }
            cout << endl;
        }
    }
    cout << endl;
    for(int i = 0;i < ((height - filter_height + 2 * padding) / stride + 1);i++){
        for(int j = 0;j < ((width - filter_width + 2 * padding) / stride + 1);j++){
            cout << res[i * ((width - filter_width + 2 * padding) / stride + 1) + j] << ' ';
        }
        cout << endl;
    }*/
    for(int i = 0;i < channel;i++){
        hipFree(d_Mat[i]);
    }

    for(int i = 0;i < channel;i++){
        hipFree(d_filter[i]);
    }
    for(int k = 0;k < channel;k++){
        free(Mat[k]);
    }
    for(int i = 0;i < channel;i++){
        free(filter[i]);
    }
    hipFree(final_res);
    free(res);
}