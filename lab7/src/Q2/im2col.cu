
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
using namespace std;
#define IDX2C(i,j,ld) (((i)*(ld))+(j))

__global__ void load(float * mat,int channel_id, int channel_count, float * unroll,
                    int height_stride,int width_stride,
                    int mat_height,int mat_width,
                    int filter_height,int filter_width,
                    int res_height,int res_width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < res_height && j < res_width){
        for(int x = 0;x < filter_height;x++){
            for(int y = 0;y < filter_width;y++){
                unroll[IDX2C(IDX2C(i,j,res_width),IDX2C(x,y,filter_width) + channel_id * filter_height * filter_width ,channel_count * filter_height * filter_width)] = mat[IDX2C(i * height_stride + x,j * width_stride + y,mat_width)];
            }
        }
    }
    __syncthreads();
}

__global__ void MatMul(float * A, float * B ,float * C,int m,int n,int k)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;
    if (i < m && j < k){
        for(int x = 0;x < n;x++){
            sum += A[i * n + x] * B[x * k + j];
        }
        C[i * k + j] = sum;
    }
}

int main()
{
    timeval t1, t2;
    int x,y;    
    cout << "Input threadsPerBlock.x:";
    cin >> x;
    cout << "Input threadsPerBlock.y:";
    cin >> y;
    dim3 threadsPerBlock(x,y);
    int height, width;
    cout << "Input problem size:";
    cin >> height;
    width = height;
    int filter_height = 3;
    int filter_width = 3;
    
    int channel = 3;
    int stride;
    cout << "Input stride:";
    cin >> stride;
    int padding = ((((height - filter_height) / stride + 1) * stride - (height - filter_height)) % stride ) / 2;

    float* Mat[channel];
    float* filter[channel];

    int Mat_size = sizeof(float) * (height + 2 * padding) * (width + 2 * padding);
    int res_size = sizeof(float) * ((height - filter_height + 2 * padding) / stride + 1) * ((width - filter_width + 2 * padding) / stride + 1);
    int filter_size = sizeof(float) * filter_height * filter_width;

    for(int k = 0;k < channel;k++){
        Mat[k] = (float*) malloc(Mat_size);
        memset(Mat[k],0,sizeof(Mat[k]));
        for(int i = padding;i < height + padding;i++){
            for(int j = padding;j < width + padding;j++){
                Mat[k][IDX2C(i,j,width + 2 * padding)] = IDX2C(i,j,width + 2 * padding);
            }
        }
    }
    for(int i = 0;i < channel;i++){
        filter[i] = (float*)malloc(filter_size);
        for(int j = 0;j < filter_height * filter_width;j++){
            filter[i][j] = j + 1;
        }
    }
    float * res;
    res = (float*)malloc(res_size);

    gettimeofday(&t1, NULL);
    float * d_Mat[channel], *d_filter[channel], *d_res;

    for(int i = 0;i < channel;i++){
        hipMalloc(&d_Mat[i], Mat_size);
        hipMemcpy(d_Mat[i],Mat[i],Mat_size,hipMemcpyHostToDevice);
    }

    for(int i = 0;i < channel;i++){
        hipMalloc(&d_filter[i],filter_size);
        hipMemcpy(d_filter[i],filter[i],filter_size,hipMemcpyHostToDevice);
    }
    hipMalloc(&d_res, res_size);
    
    float * unroll;
    hipMalloc(&unroll,res_size * filter_height * filter_width * channel );

    int res_height = (height - filter_height + 2 * padding) / stride + 1;
    int res_width = (width - filter_width + 2 * padding) / stride + 1;
    dim3 numBlocks((res_height % threadsPerBlock.x) ? res_height / threadsPerBlock.x + 1 : res_height / threadsPerBlock.x ,(res_width % threadsPerBlock.y) ? res_width / threadsPerBlock.y + 1 : res_width / threadsPerBlock.y);
        
    for(int i = 0;i < channel;i++){
        load<<<numBlocks, threadsPerBlock>>>(d_Mat[i],i,channel, unroll,stride,stride,height + 2 * padding,width + 2 * padding,filter_height,filter_width,res_height,res_width);
    }

    float * W;
    hipMalloc(&W,filter_size * channel);

    for(int i = 0;i < channel;i++){
        hipMemcpy(W + i * (filter_height * filter_width),filter[i],filter_size,hipMemcpyHostToDevice);
    }

    dim3 numBlocks1((res_height * res_width % threadsPerBlock.x) ? res_height * res_width / threadsPerBlock.x + 1 :res_height * res_width / threadsPerBlock.x ,(filter_height * filter_width % threadsPerBlock.y) ? filter_height * filter_width / threadsPerBlock.y + 1 : filter_height * filter_width / threadsPerBlock.y);

    MatMul<<<numBlocks1,threadsPerBlock>>>(unroll,W,d_res,res_height*res_width,filter_height * filter_width * channel, 1);
    gettimeofday(&t2, NULL);
    printf("convolution time is:%ldμs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);

    hipMemcpy(res, d_res, res_size, hipMemcpyDeviceToHost);


/*
    for(int x = 0;x < channel;x++){
        cout << "Input channel " << x << " after padding:" << endl;
        for(int i = 0;i < height + 2 * padding;i++){
            for(int j = 0;j < width + 2 * padding;j++){
                cout << Mat[x][i * (width + 2 * padding) + j] << " ";
            }
            cout << endl;
        }
    }

    cout << endl;

    for(int x = 0;x < channel;x++){
        cout << "Filter channel " << x << ":" << endl;
        for(int i = 0;i < filter_height;i++){
            for(int j = 0;j < filter_width;j++){
                cout << filter[x][i * filter_width + j] << ' ';
            }
            cout << endl;
        }
    }
    cout << endl;
    cout << "Res:" << endl;
    for(int i = 0;i < ((height - filter_height + 2 * padding) / stride + 1);i++){
        for(int j = 0;j < ((width - filter_width + 2 * padding) / stride + 1);j++){
            cout << res[i * ((width - filter_width + 2 * padding) / stride + 1) + j] << ' ';
        }
        cout << endl;
    }
*/
    for(int i = 0;i < channel;i++){
        hipFree(d_Mat[i]);
    }

    for(int i = 0;i < channel;i++){
        hipFree(d_filter[i]);
    }
    hipFree(d_res);
    for(int k = 0;k < channel;k++){
        free(Mat[k]);
    }
    for(int i = 0;i < channel;i++){
        free(filter[i]);
    }
    free(res);
}