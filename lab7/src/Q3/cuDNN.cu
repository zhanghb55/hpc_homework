#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hipDNN.h>
 
#define IDX2C(i,j,ld) (((i)*(ld))+(j))
using namespace std;
#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    cout \
        << "    Error occurred: " << err << endl; \
    exit(1); \
  } \
}
 
#define CUDNN_CALL(f) { \
  hipdnnStatus_t err = (f); \
  if (err != HIPDNN_STATUS_SUCCESS) { \
    cout \
        << "    Error occurred: " << err << endl; \
    exit(1); \
  } \
}
 
__global__ void assignment(float *px) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid;
}
 
void print(const float *data, int n, int c, int h, int w) {
  vector<float> buffer(1 << 20);
  CUDA_CALL(hipMemcpy(
        buffer.data(), data,
        n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
  int a = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < c; ++j) {
      cout << "n=" << i << ", c=" << j << ":" << endl;
      for (int k = 0; k < h; ++k) {
        for (int l = 0; l < w; ++l) {
          cout << setw(4) << right << buffer[a];
          ++a;
        }
        cout << endl;
      }
    }
  }
  cout << endl;
}
 
int main() {
    timeval t1, t2;
    int height, width;
    cout << "Input problem size:";
    cin >> height;
    width = height;
    int filter_height = 3;
    int filter_width = 3;

    int channel = 3;
    int stride;
    cout << "Input stride:";
    cin >> stride;
    int padding = ((((height - filter_height) / stride + 1) * stride - (height - filter_height)) % stride ) / 2;

    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));

    float* Mat;
    float* filter;
    int Mat_size = sizeof(float) * (height + 2 * padding) * (width + 2 * padding);
    int res_size = sizeof(float) * ((height - filter_height + 2 * padding) / stride + 1) * ((width - filter_width + 2 * padding) / stride + 1);
    int filter_size = sizeof(float) * filter_height * filter_width;


    int res_height = (height - filter_height + 2 * padding) / stride + 1;
    int res_width = (width - filter_width + 2 * padding) / stride + 1;


    Mat = (float*) malloc(Mat_size * channel);
    memset(Mat,0,sizeof(Mat));
    for(int k = 0;k < channel;k++){
      for(int i = padding;i < height + padding;i++){
          for(int j = padding;j < width + padding;j++){
              Mat[k * (height + 2 * padding) * (width + 2 * padding) + IDX2C(i,j,width + 2 * padding)] = IDX2C(i,j,width + 2 * padding);
          }
      }
    }

    filter = (float*)malloc(filter_size * channel);
    for(int i = 0;i < channel;i++){
        for(int j = 0;j < filter_height * filter_width;j++){
            filter[i * filter_height * filter_width + j] = j + 1;
        }
    }
    gettimeofday(&t1, NULL);
    const int in_n = 1;
    const int in_c = 3;
    const int in_h = height;
    const int in_w = width;

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
          in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
          in_n, in_c, in_h, in_w));

    float *in_data;
    CUDA_CALL(hipMalloc(
          &in_data, in_n * in_c * in_h * in_w * sizeof(float)));

    // filter
    const int filt_k = 1;
    const int filt_c = 3;
    const int filt_h = filter_height;
    const int filt_w = filter_width;


    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
          filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
          filt_k, filt_c, filt_h, filt_w));
    
    float *filt_data;
    CUDA_CALL(hipMalloc(
        &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));

    const int pad_h = padding;
    const int pad_w = padding;
    const int str_h = stride;
    const int str_w = stride;
    const int dil_h = 1;
    const int dil_w = 1;

    float * res;
    res = (float*)malloc(res_size);
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
          conv_desc,
          pad_h, pad_w, str_h, str_w, dil_h, dil_w,
          HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // output
    int out_n;
    int out_c;
    int out_h;
    int out_w;
      
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
          conv_desc, in_desc, filt_desc,
          &out_n, &out_c, &out_h, &out_w));


    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
          out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
          out_n, out_c, out_h, out_w));

    float *out_data;
    CUDA_CALL(hipMalloc(
          &out_data, out_n * out_c * out_h * out_w * sizeof(float)));


    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
          cudnn,
          in_desc, filt_desc, conv_desc, out_desc,
          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

    cout << "Convolution algorithm: " << algo << endl;
    cout << endl;


    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
          cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

    float *ws_data;
    CUDA_CALL(hipMalloc(&ws_data, ws_size));

    cout << "Workspace size: " << ws_size << endl;
    cout << endl;


    float alpha = 1.f;
    float beta = 0.f;


    assignment<<<in_w * in_h, in_n * in_c>>>(in_data);
    hipMemcpy(filt_data,filter,filt_k * filt_c * filt_h * filt_w * sizeof(float),hipMemcpyHostToDevice);
    
    
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo, ws_data, ws_size,
        &beta, out_desc, out_data));
    
    gettimeofday(&t2, NULL);
    printf("cuDNN time is:%ld μs\n",t2.tv_sec*1000000 + t2.tv_usec - t1.tv_sec*1000000 - t1.tv_usec);
/*
     cout << "in_data:" << endl;
     print(in_data, in_n, in_c, in_h, in_w);
      
     cout << "filt_data:" << endl;
     print(filt_data, filt_k, filt_c, filt_h, filt_w);
      
    cout << "res:" << endl;

    hipMemcpy(res,out_data,res_size,hipMemcpyDeviceToHost);

    for(int i = 0;i < ((height - filter_height + 2 * padding) / stride + 1);i++){
      for(int j = 0;j < ((width - filter_width + 2 * padding) / stride + 1);j++){
          cout << res[i * ((width - filter_width + 2 * padding) / stride + 1) + j] << ' ';
      }
      cout << endl;
    }*/

    CUDA_CALL(hipFree(ws_data));
    CUDA_CALL(hipFree(out_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDA_CALL(hipFree(filt_data));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDA_CALL(hipFree(in_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));
    return 0;
}